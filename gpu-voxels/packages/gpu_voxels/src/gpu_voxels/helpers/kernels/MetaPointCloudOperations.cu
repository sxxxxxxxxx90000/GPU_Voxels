#include "hip/hip_runtime.h"
// this is for emacs file handling -*- mode: c++; indent-tabs-mode: nil -*-

// -- BEGIN LICENSE BLOCK ----------------------------------------------
// This file is part of the GPU Voxels Software Library.
//
// This program is free software licensed under the CDDL
// (COMMON DEVELOPMENT AND DISTRIBUTION LICENSE Version 1.0).
// You can find a copy of this license in LICENSE.txt in the top
// directory of the source code.
//
// © Copyright 2014 FZI Forschungszentrum Informatik, Karlsruhe, Germany
//
// -- END LICENSE BLOCK ------------------------------------------------


//----------------------------------------------------------------------
/*!\file
 *
 * \author  Andreas Hermann
 * \date    2014-06-17
 *
 * MetaPointCloud kernel calls
 */
//----------------------------------------------------------------------

#include "MetaPointCloudOperations.h"

namespace gpu_voxels {

__global__
void kernelDebugMetaPointCloud(MetaPointCloudStruct* meta_point_clouds_struct)
{

  printf("================== kernelDebugMetaPointCloud DBG ================== \n");


  const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i == 0)
  {
    printf("kernelDebugMetaPointCloud DBG: NumClouds: %d \n",
           meta_point_clouds_struct->num_clouds);

    printf("kernelDebugMetaPointCloud DBG: m_dev_ptr_to_clouds_base_addresses: %p \n",
            meta_point_clouds_struct->clouds_base_addresses);

    for(int i = 0; i < meta_point_clouds_struct->num_clouds; i++)
    {
        printf("kernelDebugMetaPointCloud DBG: CloudSize[%d]: %d, clouds_base_addresses[%d]: %p \n",
               i, meta_point_clouds_struct->cloud_sizes[i],
               i, meta_point_clouds_struct->clouds_base_addresses[i]);

        if (meta_point_clouds_struct->cloud_sizes[i] > 0)
        {
          Vector3f min_xyz = meta_point_clouds_struct->clouds_base_addresses[i][0];
          Vector3f max_xyz = meta_point_clouds_struct->clouds_base_addresses[i][0];
          for (uint32_t j = 1; j < meta_point_clouds_struct->cloud_sizes[i]; j++)
          {
            min_xyz.x = min(min_xyz.x, meta_point_clouds_struct->clouds_base_addresses[i][j].x);
            min_xyz.y = min(min_xyz.y, meta_point_clouds_struct->clouds_base_addresses[i][j].y);
            min_xyz.z = min(min_xyz.z, meta_point_clouds_struct->clouds_base_addresses[i][j].z);

            max_xyz.x = max(max_xyz.x, meta_point_clouds_struct->clouds_base_addresses[i][j].x);
            max_xyz.y = max(max_xyz.y, meta_point_clouds_struct->clouds_base_addresses[i][j].y);
            max_xyz.z = max(max_xyz.z, meta_point_clouds_struct->clouds_base_addresses[i][j].z);
          }

          printf("kernelDebugMetaPointCloud DBG: CloudSize[%d] bounds: Min[%f, %f, %f], Max[%f, %f, %f] \n",
                 i, min_xyz.x, min_xyz.y, min_xyz.z, max_xyz.x, max_xyz.y, max_xyz.z);
        }
    }
  }

  printf("================== END kernelDebugMetaPointCloud DBG ================== \n");
}


__global__
void kernelTransformCloud(const Matrix4f *transformation, const Vector3f *startAddress, Vector3f *transformedAddress, uint32_t numberOfPoints)
{
  // copying the transformation matrix to local memory might be faster than accessing it from the global memory
  Matrix4f transform;
  transform = *transformation;

  uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

  while(i < numberOfPoints)
  {
    transformedAddress[i] = transform * startAddress[i];
    i += blockDim.x * gridDim.x;
  }
}

__global__
void kernelScaleCloud(const Vector3f scaling, const Vector3f* startAddress, Vector3f* transformedAddress, uint32_t numberOfPoints)
{
  uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

  while(i < numberOfPoints)
  {
    transformedAddress[i] = scaling * startAddress[i];
    i += blockDim.x * gridDim.x;
  }
}


} // end of namespace gpu_voxels
